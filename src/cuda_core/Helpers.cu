//
// Created by Jlisowskyy on 15/11/24.
//

#include "Helpers.cuh"
#include "cuda_Board.cuh"

#include <iostream>
#include <format>
#include <thrust/pair.h>

void AssertSuccess(hipError_t error, const char *file, int line) {
    TraceError(error, file, line);

    if (error != hipSuccess) {
        exit(EXIT_FAILURE);
    }
}

bool TraceError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        std::cerr << std::format("CUDA Error at {}:{} - {}\n", file, line, hipGetErrorString(error)) << std::endl;
    }

    return error != hipSuccess;
}

HYBRID static thrust::pair<char, char> ConvertToCharPos(const __uint32_t boardPosMsb) {
    const __uint32_t boardPos = ConvertToReversedPos(boardPosMsb);
    return {static_cast<char>('a' + (boardPos % 8)), static_cast<char>('1' + (boardPos / 8))};
}

__device__ const char IndexToFigCharMap[BIT_BOARDS_COUNT]{
        'P', 'N', 'B', 'R', 'Q', 'K', 'p', 'n', 'b', 'r', 'q', 'k',
};

HYBRID void DisplayBoard(const cuda_Board *board) {
    static constexpr __uint32_t LastRowIndex = 7;
    static constexpr __uint32_t CharsForFig = 3;
    static constexpr __uint32_t FigsPerRow = 8;
    static constexpr __uint32_t FigsPerCol = 8;

    for (__uint32_t y = 0; y < FigsPerCol; ++y) {
        for (__uint32_t x = 0; x < FigsPerRow; ++x) {
            const uint64_t field = 1LLU << (y*8 + x);
            printf(" ");

            bool found = false;
            for (__uint32_t desc = 0; desc < BIT_BOARDS_COUNT; ++desc) {
                if ((board->BitBoards[desc] & field) != 0) {
                    printf("%c", IndexToFigCharMap[desc]);
                    found = true;
                    break;
                }
            }

            if (!found) {
                printf(" ");
            }
            printf(" ");

            if (x != LastRowIndex) {
                printf("|");
            } else {
                printf("   %c", static_cast<char>('8' - y));
            }
        }

        printf("\n");
        if (y != LastRowIndex) {
            for (__uint32_t idx = 0; idx < LastRowIndex + CharsForFig * FigsPerRow; ++idx) {
                printf("-");
            }
            printf("\n");
        }
    }

    for (__uint32_t idx = 0; idx < LastRowIndex + CharsForFig * FigsPerRow; ++idx) {
        printf("-");
    }
    printf("\n");

    for (__uint32_t x = 0; x < FigsPerRow; ++x) {
        printf(" %c  ", static_cast<char>('A' + x));
    }
    printf("\n");

    printf("Moving color: %s\n", board->MovingColor == WHITE ? "white" : "black");
    printf("Possible castlings:\n");
    static constexpr const char *castlingNames[] = {
            "White King Side", "White Queen Side", "Black King Side", "Black Queen Side"
    };

    for (__uint32_t i = 0; i < CASTLING_COUNT; ++i) {
        printf("%s: %d\n", castlingNames[i], board->GetCastlingRight(i));
    }


    const __uint32_t msbPos = ExtractMsbPosNeutral(board->ElPassantField);
    auto [c1, c2] = ConvertToCharPos(msbPos);
    char str[] = {c1, c2, '\0'};

    printf("El passant field: %s\n",
           (board->ElPassantField == INVALID_EL_PASSANT_BIT_BOARD ? "-" : str));
}
