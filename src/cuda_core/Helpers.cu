//
// Created by Jlisowskyy on 15/11/24.
//

#include "Helpers.cuh"

#include <iostream>
#include <format>

void AssertSuccess(hipError_t error, const char *file, int line) {
    TraceError(error, file, line);

    if (error != hipSuccess) {
        exit(EXIT_FAILURE);
    }
}

bool TraceError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        std::cerr << std::format("CUDA Error at {}:{} - {}\n", file, line, hipGetErrorString(error)) << std::endl;
    }

    return error != hipSuccess;
}
